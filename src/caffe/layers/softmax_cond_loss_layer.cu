#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_cond_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void SoftmaxWithCondLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
    cond_ = static_cast<int>(bottom[2]->cpu_data()[0]);
  if(cond_==0){
  SoftmaxWithLossLayer<Dtype>::Forward_gpu(bottom,top);
  }
  else{
    top[0]->mutable_cpu_data()[0] = 0;
  }
}

template <typename Dtype>
__global__ void SoftmaxCondLossBackwardGPU(const int nthreads, Dtype* bottom_diff, 
                const int num, const int dim, const int spatial_dim) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
  }
}

template <typename Dtype>
void SoftmaxWithCondLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    //backpropagate when cond_==0
    if(cond_==0)
      SoftmaxWithLossLayer<Dtype>::Backward_cpu(top,propagate_down,bottom);
    else{
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      // const Dtype* label = bottom[1]->cpu_data();
      int dim =  SoftmaxWithLossLayer<Dtype>::prob_.count() / SoftmaxWithLossLayer<Dtype>::outer_num_;
      const int nthreads = SoftmaxWithLossLayer<Dtype>::outer_num_ * SoftmaxWithLossLayer<Dtype>::inner_num_;
      // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxCondLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom_diff,
        SoftmaxWithLossLayer<Dtype>::outer_num_, dim, SoftmaxWithLossLayer<Dtype>::inner_num_);

    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithCondLossLayer);

}  // namespace caffe
